#include "hip/hip_runtime.h"
// spread_functions.cuh y otras dependencias asumidas cargadas correctamente
#include "spread_functions.cuh"

#define _USE_MATH_DEFINES
#include <cmath>
#include <vector>
#include <omp.h>
#include <iostream>
#include <array>
#include <random>

#include "fires.hpp"
#include "landscape.hpp"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

struct DeviceBuffers {
    int* frontier_0;
    int* frontier_1;
    int* next_frontier_0;
    int* next_frontier_1;
    int* frontier_size;
    int* next_frontier_count;
    int* done_flag;
    int* burned_bin;
    int* iteration_map;
    unsigned int* processed_cells;

    float* elevation;
    float* fwi;
    float* aspect;
    float* wind_dir;
    float* vegetation_type;
    uint8_t* burnable;

    SimulationParams* d_params;
    hiprandState* rng_states;
};

struct FireKernelParams {
    const float* elevation;
    const float* fwi;
    const float* aspect;
    const float* wind_dir;
    const float* vegetation_type;
    const uint8_t* burnable;

    int* burned_bin;
    int width;
    int height;

    unsigned int* processed_cells;
    const SimulationParams* params;

    float distance;
    float upper_limit;
    float elevation_mean;
    float elevation_sd;
};

constexpr float PIf = 3.1415927f;
constexpr float h_angles[8] = {
    PIf * 3 / 4, PIf, PIf * 5 / 4, PIf / 2,
    PIf * 3 / 2, PIf / 4, 0, PIf * 7 / 4
};
constexpr int h_moves[8][2] = {
    { -1, -1 }, { -1, 0 }, { -1, 1 }, { 0, -1 },
    { 0, 1 }, { 1, -1 }, { 1, 0 }, { 1, 1 }
};
__constant__ float d_angles[8];
__constant__ int d_moves[8][2];


////////////////////////////////// DEVICE //////////////////////////////


__global__ void init_rng_kernel(hiprandState* states, int width, int height, int seed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total = width * height;
    if (tid < total) {
        int i = tid % width;
        int j = tid / width;
        unsigned long long cell_seed = seed ^ (i * 73856093) ^ (j * 19349663);
        hiprand_init(cell_seed, 0, 0, &states[tid]);
    }
}


__device__ void spread_probability(
    float burning_elevation,
    float burning_wind_direction,
    const float* elevations,
    const float* vegetation_types,
    const float* fwis,
    const float* aspects,
    const float* upper_limits,
    const SimulationParams* params,
    float distance,
    float elevation_mean,
    float elevation_sd,
    float* probs_out
) {
    float fwi_pred = params->fwi_pred;
    float aspect_pred = params->aspect_pred;
    float wind_pred = params->wind_pred;
    float elevation_pred = params->elevation_pred;
    float slope_pred = params->slope_pred;
    float independent_pred = params->independent_pred;
    for (int n = 0; n < 8; n++) {
        float slope_term = __sinf(atanf((elevations[n] - burning_elevation) / distance));
        float wind_term = __cosf(d_angles[n] - burning_wind_direction);
        float elev_term = (elevations[n] - elevation_mean) / elevation_sd;

        float linpred = independent_pred;

        if ((int)vegetation_types[n] == SUBALPINE) {
            linpred += params->subalpine_pred;
        } else if ((int)vegetation_types[n] == WET) {
            linpred += params->wet_pred;
        } else if ((int)vegetation_types[n] == DRY) {
            linpred += params->dry_pred;
        }

        linpred += fwi_pred * fwis[n];
        linpred += aspect_pred * aspects[n];
        linpred += wind_term * wind_pred + elev_term * elevation_pred + slope_term * slope_pred;

        probs_out[n] = upper_limits[n] / (1.0f + __expf(-linpred));
    }
}


__global__ void fire_persistent_kernel(
    FireKernelParams args,
    int* frontier_0, int* frontier_1,
    int* frontier_size,
    int* next_frontier_0, int* next_frontier_1,
    int* next_frontier_count,
    int* iteration_map,
    int iteration_tag,
    int* done_flag,
    hiprandState* rng_states
) {
    unsigned long long start = clock64();
    const float* elevation = args.elevation;
    const float* fwi = args.fwi;
    const float* aspect = args.aspect;
    const float* wind_dir = args.wind_dir;
    const float* vegetation_type = args.vegetation_type;
    const uint8_t* burnable = args.burnable;

    int* burned_bin = args.burned_bin;
    int width = args.width;
    int height = args.height;
    const SimulationParams* params = args.params;

    float distance = args.distance;
    float upper_limit = args.upper_limit;
    float elevation_mean = args.elevation_mean;
    float elevation_sd = args.elevation_sd;
    
    unsigned int local_processed_cells = 0;
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    while (!*done_flag) {
        int frontier_len = *frontier_size;

        for (int idx = tid; idx < frontier_len; idx += gridDim.x * blockDim.x) {
            int i = frontier_0[idx];
            int j = frontier_1[idx];
            int center_idx = j * width + i;

            hiprandState local_state = rng_states[center_idx];

            float elev_c = elevation[center_idx];
            float wind_c = wind_dir[center_idx];

            int n_coords_0[8], n_coords_1[8];
            int n_indices[8];
            uint8_t n_out_flags[8];
            float n_elev[8], n_fwi[8], n_asp[8], n_veg[8], n_burn[8], n_upper[8];

            for (int n = 0; n < 8; ++n) {
                int ni = i + d_moves[n][0];
                int nj = j + d_moves[n][1];
                n_coords_0[n] = ni;
                n_coords_1[n] = nj;

                if (ni < 0 || nj < 0 || ni >= width || nj >= height) {
                    n_out_flags[n] = 1;
                    n_indices[n] = 0;
                    n_elev[n] = n_fwi[n] = n_asp[n] = n_veg[n] = 0.0f;
                    n_burn[n] = 0;
                } else {
                    int n_idx = nj * width + ni;
                    n_out_flags[n] = 0;
                    n_indices[n] = n_idx;
                    n_elev[n] = elevation[n_idx];
                    n_fwi[n] = fwi[n_idx];
                    n_asp[n] = aspect[n_idx];
                    n_veg[n] = vegetation_type[n_idx];
                    n_burn[n] = burnable[n_idx];
                    if (!n_out_flags[n]) {
                        ++local_processed_cells;
                    }
                }

                uint8_t burnable_mask = (!burned_bin[n_indices[n]] && n_burn[n]);
                uint8_t valid_mask = !n_out_flags[n] && burnable_mask;
                n_upper[n] = valid_mask * upper_limit;
            }

            float n_probs[8];
            spread_probability(
                elev_c, wind_c,
                n_elev, n_veg, n_fwi, n_asp, n_upper,
                params, distance, elevation_mean, elevation_sd,
                n_probs
            );

            for (int n = 0; n < 8; ++n) {
                float rnd = hiprand_uniform(&local_state);
                if (rnd < n_probs[n]) {
                    if (n_indices[n] >= 0 && !n_out_flags[n] && n_burn[n]) {
                        if (atomicCAS(&iteration_map[n_indices[n]], 0, iteration_tag) == 0) {
                            burned_bin[n_indices[n]] = 1;
                            int pos = atomicAdd(next_frontier_count, 1);
                            next_frontier_0[pos] = n_coords_0[n];
                            next_frontier_1[pos] = n_coords_1[n];
                        }
                    }
                }
            }
            rng_states[center_idx] = local_state;
        }

        __syncthreads();

        if (tid == 0) {
            int count = *next_frontier_count;
            *frontier_size = count;
            *next_frontier_count = 0;
            *done_flag = (count == 0);
        }

        __syncthreads();

        int* tmp0 = frontier_0;
        int* tmp1 = frontier_1;
        frontier_0 = next_frontier_0;
        frontier_1 = next_frontier_1;
        next_frontier_0 = tmp0;
        next_frontier_1 = tmp1;
    }

    if (local_processed_cells)
        atomicAdd(args.processed_cells, local_processed_cells);
    unsigned long long end = clock64();
}


////////////////////////////// HOST //////////////////////////////


DeviceBuffers allocate_device_memory(size_t MAX_CELLS) {
    DeviceBuffers buf = {};
    hipMalloc(&buf.frontier_0, MAX_CELLS * sizeof(int));
    hipMalloc(&buf.frontier_1, MAX_CELLS * sizeof(int));
    hipMalloc(&buf.next_frontier_0, MAX_CELLS * sizeof(int));
    hipMalloc(&buf.next_frontier_1, MAX_CELLS * sizeof(int));
    hipMalloc(&buf.frontier_size, sizeof(int));
    hipMalloc(&buf.next_frontier_count, sizeof(int));
    hipMalloc(&buf.done_flag, sizeof(int));
    hipMalloc(&buf.burned_bin, MAX_CELLS * sizeof(int));
    hipMalloc(&buf.processed_cells, sizeof(unsigned int));
    hipMalloc(&buf.iteration_map, MAX_CELLS * sizeof(int));
    hipMemset(buf.iteration_map, 0, MAX_CELLS * sizeof(int));

    hipMalloc(&buf.elevation, MAX_CELLS * sizeof(float));
    hipMalloc(&buf.fwi, MAX_CELLS * sizeof(float));
    hipMalloc(&buf.aspect, MAX_CELLS * sizeof(float));
    hipMalloc(&buf.wind_dir, MAX_CELLS * sizeof(float));
    hipMalloc(&buf.vegetation_type, MAX_CELLS * sizeof(float));
    hipMalloc(&buf.burnable, MAX_CELLS * sizeof(uint8_t));

    hipMalloc(&buf.d_params, sizeof(SimulationParams));
    hipMalloc(&buf.rng_states, MAX_CELLS * sizeof(hiprandState));

    return buf;
}


void copy_inputs_to_device(
    const LandscapeSoA& landscape,
    const std::vector<std::pair<size_t, size_t>>& ignition_cells,
    const SimulationParams& params,
    DeviceBuffers& buf,
    int n_col,
    size_t MAX_CELLS
) {
    // Convert ignition to burned_bin
    std::vector<int> burned_bin(MAX_CELLS, 0);
    std::vector<int> h_frontier_0(MAX_CELLS, -1);
    std::vector<int> h_frontier_1(MAX_CELLS, -1);

    for (size_t i = 0; i < ignition_cells.size(); ++i) {
        auto [x, y] = ignition_cells[i];
        h_frontier_0[i] = x;
        h_frontier_1[i] = y;
        burned_bin[utils::INDEX(x, y, n_col)] = 1;
    }

    int init_size = ignition_cells.size();

    hipMemcpy(buf.frontier_0, h_frontier_0.data(), init_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(buf.frontier_1, h_frontier_1.data(), init_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(buf.frontier_size, &init_size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(buf.burned_bin, burned_bin.data(), MAX_CELLS * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(buf.elevation, landscape.elevation.data(), MAX_CELLS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(buf.fwi, landscape.fwi.data(), MAX_CELLS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(buf.aspect, landscape.aspect.data(), MAX_CELLS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(buf.wind_dir, landscape.wind_dir.data(), MAX_CELLS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(buf.vegetation_type, landscape.vegetation_type.data(), MAX_CELLS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(buf.burnable, landscape.burnable.data(), MAX_CELLS * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(buf.d_params, &params, sizeof(SimulationParams), hipMemcpyHostToDevice);

    hipMemset(buf.next_frontier_count, 0, sizeof(int));
    hipMemset(buf.done_flag, 0, sizeof(int));
    hipMemset(buf.processed_cells, 0, sizeof(unsigned int));
}


void initialize_rng(DeviceBuffers& buf, int n_row, int n_col, int seed, int threads_per_block, int num_blocks) {
    init_rng_kernel<<<num_blocks, threads_per_block>>>(buf.rng_states, n_col, n_row, seed);
}


void launch_kernel(DeviceBuffers& buf, FireKernelParams& args, int threads_per_block, int num_blocks) {
    int iteration_tag = 1;
    fire_persistent_kernel<<<num_blocks, threads_per_block>>>(
        args,
        buf.frontier_0, buf.frontier_1, buf.frontier_size,
        buf.next_frontier_0, buf.next_frontier_1, buf.next_frontier_count,
        buf.iteration_map, iteration_tag,
        buf.done_flag, buf.rng_states
    );
    hipDeviceSynchronize();
}


Fire copy_results_from_device(
    const DeviceBuffers& buf,
    size_t n_row,
    size_t n_col
) {
    size_t MAX_CELLS = n_row * n_col;

    std::vector<int> burned_bin(MAX_CELLS);
    hipMemcpy(burned_bin.data(), buf.burned_bin, MAX_CELLS * sizeof(int), hipMemcpyDeviceToHost);

    std::vector<size_t> ids_0, ids_1;
    for (size_t j = 0; j < n_row; ++j) {
        for (size_t i = 0; i < n_col; ++i) {
            if (burned_bin[utils::INDEX(i, j, n_col)]) {
                ids_0.push_back(i);
                ids_1.push_back(j);
            }
        }
    }

    unsigned int processed_cells;
    hipMemcpy(&processed_cells, buf.processed_cells, sizeof(unsigned int), hipMemcpyDeviceToHost);

    return Fire{
        n_col, n_row,
        processed_cells,
        0.0,
        burned_bin,
        ids_0, ids_1,
        { ids_0.size() }
    };
}


void free_device_memory(DeviceBuffers& buf) {
    hipFree(buf.frontier_0); hipFree(buf.frontier_1);
    hipFree(buf.next_frontier_0); hipFree(buf.next_frontier_1);
    hipFree(buf.frontier_size); hipFree(buf.next_frontier_count);
    hipFree(buf.done_flag); hipFree(buf.burned_bin);
    hipFree(buf.iteration_map); hipFree(buf.processed_cells);

    hipFree(buf.elevation); hipFree(buf.fwi); hipFree(buf.aspect);
    hipFree(buf.wind_dir); hipFree(buf.vegetation_type); hipFree(buf.burnable);

    hipFree(buf.d_params); hipFree(buf.rng_states);
}


Fire simulate_fire(
    LandscapeSoA landscape,
    size_t n_row, size_t n_col,
    const std::vector<std::pair<size_t, size_t>>& ignition_cells,
    SimulationParams params,
    float distance,
    float elevation_mean,
    float elevation_sd,
    int n_replicate,
    float upper_limit = 1.0f
) {
    const size_t MAX_CELLS = n_row * n_col;
    const int threads_per_block = 256;
    const int num_blocks = (MAX_CELLS + threads_per_block - 1) / threads_per_block;

    hipMemcpyToSymbol(HIP_SYMBOL(d_angles), h_angles, sizeof(h_angles));
    hipMemcpyToSymbol(HIP_SYMBOL(d_moves), h_moves, sizeof(h_moves));

    DeviceBuffers buf = allocate_device_memory(MAX_CELLS);

    copy_inputs_to_device(landscape, ignition_cells, params, buf, n_col, MAX_CELLS);

    initialize_rng(buf, n_row, n_col, 123 + n_replicate, threads_per_block, num_blocks);

    FireKernelParams args = {
        buf.elevation, buf.fwi, buf.aspect, buf.wind_dir, buf.vegetation_type,
        buf.burnable, buf.burned_bin,
        static_cast<int>(n_col), static_cast<int>(n_row),
        buf.processed_cells,
        buf.d_params,
        distance, upper_limit, elevation_mean, elevation_sd,
    };

    // 🔽 NUEVO: MEDICIÓN DE TIEMPO CON EVENTOS CUDA
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    launch_kernel(buf, args, threads_per_block, num_blocks);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    float seconds = milliseconds / 1000.0f;

    Fire result = copy_results_from_device(buf, n_row, n_col);

    free_device_memory(buf);

    result.time_taken = seconds;

    return result;
}
